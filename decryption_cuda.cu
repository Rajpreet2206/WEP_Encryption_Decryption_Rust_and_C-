
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <cstdint>

// CUDA kernel to perform decryption on the GPU
__global__ void wep_decrypt_kernel(const char* ciphertext, size_t ciphertext_length, const uint8_t* wep_key, size_t key_length, char* decrypted_text) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < ciphertext_length; i += stride) {
        if (ciphertext[i] != ' ') {
            // Convert 8-bit binary representation to an 8-bit integer
            uint8_t encrypted_byte = 0;
            for (int j = 0; j < 8; j++) {
                if (ciphertext[i + j] == '1') {
                    encrypted_byte |= (1 << (7 - j));
                }
            }
            decrypted_text[i] = encrypted_byte ^ wep_key[i / 8 % key_length];
        }
    }
}

int countCharacters(const std::string& str) {
    return str.size();
}
template<typename Duration, typename Function, typename... Args>
Duration time_taken_by_the_function(const std::string& function_name, Function&& function, Args&&... args) {
    auto Tstart = std::chrono::high_resolution_clock::now();
    std::forward<Function>(function)(std::forward<Args>(args)...);
    auto Tend = std::chrono::high_resolution_clock::now();
    auto time_duration = std::chrono::duration_cast<Duration>(Tend - Tstart);
    std::cout << "Executing the '" << function_name << "' took " << time_duration.count() << " microseconds" << std::endl;
    return time_duration;
}
std::string wep_decrypt(const std::string& ciphertext, const std::vector<uint8_t>& wep_key) {
    std::string plaintext;
    size_t key_length = wep_key.size();
    // Allocate memory on the host
    char* host_decrypted_text = new char[ciphertext.size() + 1];
    // Allocate memory on the device (GPU)
    char* device_ciphertext;
    char* device_decrypted_text;
    uint8_t* device_wep_key;
    size_t ciphertext_size = ciphertext.size();
    hipMalloc((void**)&device_ciphertext, (ciphertext_size + 1) * sizeof(char));
    hipMalloc((void**)&device_decrypted_text, (ciphertext_size + 1) * sizeof(char));
    hipMalloc((void**)&device_wep_key, key_length * sizeof(uint8_t));
    // Copy data from host to device
    hipMemcpy(device_wep_key, wep_key.data(), key_length * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(device_ciphertext, ciphertext.c_str(), (ciphertext_size + 1) * sizeof(char), hipMemcpyHostToDevice);
    // Launch the kernel on the GPU
    int num_threads_per_block = 256;
    int num_blocks = (ciphertext_size + num_threads_per_block - 1) / num_threads_per_block;
    wep_decrypt_kernel<<<num_blocks, num_threads_per_block>>>(device_ciphertext, ciphertext_size, device_wep_key, key_length, device_decrypted_text);
    // Copy the decrypted text back to the host
    hipMemcpy(host_decrypted_text, device_decrypted_text, (ciphertext_size + 1) * sizeof(char), hipMemcpyDeviceToHost);
    host_decrypted_text[ciphertext_size] = '\0';
    // Clean up memory on the device
    hipFree(device_ciphertext);
    hipFree(device_decrypted_text);
    hipFree(device_wep_key);
    // Clean up memory on the host
    plaintext = std::string(host_decrypted_text);
    delete[] host_decrypted_text;
    return plaintext;
}
int main() {
    std::string ciphertext = "11011101 11011110 11101100 10101101 10011100 11001111 11001000 10101001 10110011 10011010 10001010 11010011 10100011 10101010 10111110 11011101 11011110 11101100 10101101 10011100 11001111 11001000 10101001 10110011 10011010 10001010 11010011 10100011 10101010 10111110\n"
    "11011101 11011110 11101100 10101101 10011100 11001111 11001000 10101001 10110011 10011010 10001010 11010011 10100011 10101010 10111110 11011101 11011110 11101100 10101101 10011100 11001111 11001000 10101001 10110011 10011010 10001010 11010011 10100011 10101010 10111110";
    std::vector<uint8_t> wep_key = { 0xAA, 0xBB, 0xCC, 0xDD, 0xEE };
    std::string decrypted_text = wep_decrypt(ciphertext, wep_key);
    int Characters_Count = countCharacters(ciphertext);
    std::cout << "Number of characters: " << Characters_Count << std::endl;
    //std::cout << "Decrypted Text: " << decrypted_text << std::endl;
    auto duration = time_taken_by_the_function<std::chrono::microseconds>("wep_decrypt", wep_decrypt, ciphertext, wep_key);
    return 0;
}
